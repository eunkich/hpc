#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cmath>

__device__ hipDoubleComplex complexToCuComplex(double real, double imag)
{
    return make_hipDoubleComplex(real, imag);
}

__device__ void cuComplexToComplex(hipDoubleComplex c, double *real, double *imag)
{
    *real = hipCreal(c);
    *imag = hipCimag(c);
}

__device__ hipDoubleComplex operator*(hipDoubleComplex a, hipDoubleComplex b)
{
    return make_hipDoubleComplex(hipCreal(a) * hipCreal(b) - hipCimag(a) * hipCimag(b),
                                hipCreal(a) * hipCimag(b) + hipCimag(a) * hipCreal(b));
}

__device__ hipDoubleComplex operator/(hipDoubleComplex a, double b)
{
    return make_hipDoubleComplex(hipCreal(a) / b, hipCimag(a) / b);
}

__global__ void computeDdx(hipDoubleComplex *wave_gpu, hipDoubleComplex *fft_3_gpu, double kx, int nxyz)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nxyz)
    {
        hipDoubleComplex factor = make_hipDoubleComplex(0.0, kx);
        fft_3_gpu[idx] = wave_gpu[idx] * factor;
    }
}

__global__ void computeScale(hipDoubleComplex *fft_3_gpu, hipDoubleComplex *d_dx_gpu, double scale, int nxyz)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nxyz)
    {
        d_dx_gpu[idx] = fft_3_gpu[idx] / scale;
    }
}

int main()
{
    // Parameters
    // ndim 16 -> 256 stride *2 | ntrial >= 3
    std::cout << "ndim,elapsed_time,num_FLOP,FLOPs" << std::endl;
    for (int ndim = 16; ndim < 512; ndim *= 2)
    {
        for (int trial = 0; trial < 10; trial++)
        {
            const int nx = ndim;   // Number of lattice points in x-direction
            const int ny = ndim;   // Number of lattice points in y-direction
            const int nz = ndim;   // Number of lattice points in z-direction
            const double lx = 1.0; // Length of the cubic lattice in x-direction
            const double ly = 1.0; // Length of the cubic lattice in y-direction
            const double lz = 1.0; // Length of the cubic lattice in z-direction

            // Compute total number of lattice points
            const int nxyz = nx * ny * nz;

            // Wave vector components in reciprocal lattice units
            const double kx = 2.0 * M_PI / lx * 2; // Wave vector component in x-direction
            const double ky = 2.0 * M_PI / ly * 3; // Wave vector component in y-direction
            const double kz = 2.0 * M_PI / lz * 4; // Wave vector component in z-direction

            // Compute distances between spatial lattice sites
            const double dx = lx / nx;
            const double dy = ly / ny;
            const double dz = lz / nz;

            // Allocate memory for wave, fft_3, d_dx, d_dy, and d_dz arrays
            hipDoubleComplex *wave = new hipDoubleComplex[nxyz];
            hipDoubleComplex *fft_3 = new hipDoubleComplex[nxyz];
            hipDoubleComplex *d_dx = new hipDoubleComplex[nxyz];
            hipDoubleComplex *d_dy = new hipDoubleComplex[nxyz];
            hipDoubleComplex *d_dz = new hipDoubleComplex[nxyz];

            // Allocate memory on the GPU for wave, fft_3, d_dx, d_dy, and d_dz arrays
            hipDoubleComplex *wave_gpu;
            hipDoubleComplex *fft_3_gpu;
            hipDoubleComplex *d_dx_gpu;
            hipDoubleComplex *d_dy_gpu;
            hipDoubleComplex *d_dz_gpu;

            hipMalloc((void **)&wave_gpu, nxyz * sizeof(hipDoubleComplex));
            hipMalloc((void **)&fft_3_gpu, nxyz * sizeof(hipDoubleComplex));
            hipMalloc((void **)&d_dx_gpu, nxyz * sizeof(hipDoubleComplex));
            hipMalloc((void **)&d_dy_gpu, nxyz * sizeof(hipDoubleComplex));
            hipMalloc((void **)&d_dz_gpu, nxyz * sizeof(hipDoubleComplex));

            // Create cuFFT plans
            hipfftHandle forward_plan;
            hipfftHandle backward_plan;
            hipfftPlan3d(&forward_plan, nx, ny, nz, HIPFFT_Z2Z);
            hipfftPlan3d(&backward_plan, nx, ny, nz, HIPFFT_Z2Z);

            // Generate complex plane wave on the cubic lattice
            for (int i = 0; i < nxyz; ++i)
            {
                const int ix = i % nx;
                const int iy = (i / nx) % ny;
                const int iz = i / (nx * ny);
                const double x = (ix - nx / 2) * dx;
                const double y = (iy - ny / 2) * dy;
                const double z = (iz - nz / 2) * dz;
                const double phase = kx * x + ky * y + kz * z;

                wave[i] = make_hipDoubleComplex(cos(phase), sin(phase));
            }

            // for CUDA kernel to compute d/dx
            int blockSize = 256;
            int numBlocks = (nxyz + blockSize - 1) / blockSize;

            // start timing here
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);

            // Copy data from CPU to GPU
            hipMemcpy(wave_gpu, wave, nxyz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

            // Perform forward FFT
            hipfftExecZ2Z(forward_plan, wave_gpu, fft_3_gpu, HIPFFT_FORWARD);

            // make a copy of the forward transform
            hipMemcpy(wave_gpu, fft_3_gpu, nxyz * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

            // Compute d/dx
            computeDdx<<<numBlocks, blockSize>>>(wave_gpu, fft_3_gpu, kx, nxyz);
            hipfftExecZ2Z(backward_plan, fft_3_gpu, fft_3_gpu, HIPFFT_BACKWARD);
            computeScale<<<numBlocks, blockSize>>>(fft_3_gpu, d_dx_gpu, static_cast<double>(nxyz), nxyz);

            // Compute d/dy
            computeDdx<<<numBlocks, blockSize>>>(wave_gpu, fft_3_gpu, ky, nxyz);
            hipfftExecZ2Z(backward_plan, fft_3_gpu, fft_3_gpu, HIPFFT_BACKWARD);
            computeScale<<<numBlocks, blockSize>>>(fft_3_gpu, d_dy_gpu, static_cast<double>(nxyz), nxyz);

            // Compute d/dz
            computeDdx<<<numBlocks, blockSize>>>(wave_gpu, fft_3_gpu, kz, nxyz);
            hipfftExecZ2Z(backward_plan, fft_3_gpu, fft_3_gpu, HIPFFT_BACKWARD);
            computeScale<<<numBlocks, blockSize>>>(fft_3_gpu, d_dz_gpu, static_cast<double>(nxyz), nxyz);

            // Copy data from GPU to CPU
            hipMemcpy(d_dx, d_dx_gpu, nxyz * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
            hipMemcpy(d_dy, d_dy_gpu, nxyz * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
            hipMemcpy(d_dz, d_dz_gpu, nxyz * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

            // end timing here
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);

            // Calculate elapsed time
            float elapsed_ms_cublas = 0;
            hipEventElapsedTime(&elapsed_ms_cublas, start, stop);

            double flop = 24 * ndim * ndim * ndim * log2(ndim) + 6 * ndim * ndim * ndim;

            std::cout << nx << "," << elapsed_ms_cublas * 1e-3f << "," << flop << ","
                      << flop / (elapsed_ms_cublas * 1e-3f) << std::endl;

            // Free GPU memory
            hipFree(wave_gpu);
            hipFree(fft_3_gpu);
            hipFree(d_dx_gpu);
            hipFree(d_dy_gpu);
            hipFree(d_dz_gpu);

            // Destroy cuFFT plans
            hipfftDestroy(forward_plan);
            hipfftDestroy(backward_plan);

            // Free CPU memory
            delete[] wave;
            delete[] d_dx;
            delete[] d_dy;
            delete[] d_dz;
        }
    }

    return 0;
}
