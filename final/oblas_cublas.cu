#include <iostream>
#include <chrono>
#include <cblas.h>        // oblas
#include <hip/hip_runtime.h> // cuda
#include <hipblas.h>    // cublas

auto cstart = std::chrono::high_resolution_clock::now();
auto cstop = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(cstop - cstart);
long double elapsed_ms_oblas = 0.L;
int main()
{
    // Print header
    std::cout << "dim,trial,ms_cuda,ms_oblas,mflops_cuda,mflops_oblas\n";

    const int MIN_DIMENSION = 16;
    const int MAX_DIMENSION = 8192;
    const int NTRIALS = 3;

    // Allocate host memory for square matrices
    double *h_A = new double[MAX_DIMENSION * MAX_DIMENSION];
    double *h_B = new double[MAX_DIMENSION * MAX_DIMENSION];
    double *h_C = new double[MAX_DIMENSION * MAX_DIMENSION];

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    for (int N = MIN_DIMENSION; N <= MAX_DIMENSION; N *= 2)
    {
        double fp_op = 0.;
        fp_op = 2. * N * N * N + 2 * N * N;

        for (int j = 0; j < NTRIALS; j++)
        { // Initialize input matrices
            for (int i = 0; i < N * N; ++i)
            {
                h_A[i] = static_cast<double>(rand()) / RAND_MAX;
                h_B[i] = static_cast<double>(rand()) / RAND_MAX;
                h_C[i] = static_cast<double>(rand()) / RAND_MAX;
            }

            // Allocate device memory for matrices
            double *d_A, *d_B, *d_C;
            hipMalloc((void **)&d_A, N * N * sizeof(double));
            hipMalloc((void **)&d_B, N * N * sizeof(double));
            hipMalloc((void **)&d_C, N * N * sizeof(double));

            // Copy input matrices from host to device
            hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_B, h_B, N * N * sizeof(double), hipMemcpyHostToDevice);

            // Perform matrix multiplication
            const double alpha = static_cast<double>(rand()) / RAND_MAX;
            const double beta = static_cast<double>(rand()) / RAND_MAX;
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start, 0);
            hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);

            // Calculate elapsed time
            float elapsed_ms_cublas = 0;
            hipEventElapsedTime(&elapsed_ms_cublas, start, stop);

            cstart = std::chrono::high_resolution_clock::now();
            cblas_dgemm(CblasColMajor, CblasNoTrans, CblasNoTrans, N, N, N, alpha, h_A, N, h_B, N, beta, h_C, N);
            cstop = std::chrono::high_resolution_clock::now();
            duration = std::chrono::duration_cast<std::chrono::nanoseconds>(cstop - cstart);
            elapsed_ms_oblas = duration.count() * 1.e-6;

            // Free device memory
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_C);

            // Print performance
            std::cout << N << "," << j << ","
                      << elapsed_ms_cublas << "," << elapsed_ms_oblas << ","
                      << fp_op / 1.e6 / elapsed_ms_cublas << "," << fp_op / 1.e6 / elapsed_ms_oblas << std::endl;
        }
    }

    // Destroy cuBLAS handle
    hipblasDestroy(handle);

    // Free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
