#include <iostream>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>
#include <hipblas.h>

auto start = std::chrono::high_resolution_clock::now();
auto stop = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
long double elapsed_time = 0.L;

int main()
{
    std::cout << "Direction,size,elapsed_time,bps" << std::endl;
    const int MIN_N = 8 / sizeof(float);
    const int MAX_N = 256 * 1e6 / sizeof(float);

    for (int N = MIN_N; N < MAX_N * 2; N *= 2)
    {
        float *h_A = new float[N];
        for (int i = 0; i < N; i++)
        {
            h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        }

        float *d_A;
        hipMalloc((void **)&d_A, N * sizeof(float));

        start = std::chrono::high_resolution_clock::now();
        hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
        elapsed_time = duration.count() * 1e-9;

        // Print performance information
        std::cout << "HostToDevice"
                  << "," << sizeof(float) * N
                  << "," << elapsed_time
                  << "," << (sizeof(float) * N) / elapsed_time
                  << std::endl;

        // Copy array from host to device
        start = std::chrono::high_resolution_clock::now();
        hipMemcpy(h_A, d_A, N * sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
        elapsed_time = duration.count() * 1e-9;
        std::cout << "DeviceToHost"
                  << "," << sizeof(float) * N
                  << "," << elapsed_time
                  << "," << (sizeof(float) * N) / elapsed_time
                  << std::endl;

        // Free device memory
        hipFree(d_A);
        // Free host memory
        delete[] h_A;
    }
}
